#include "hip/hip_runtime.h"

#include<cstdio>
#include <sys/time.h>
#include<hip/hip_runtime.h>
#include <>
#include <hip/hip_fp16.h>
// #include <immintrin.h>
// #include <avx2intrin.h>
#include "reduce.h"



static void HandleError( hipError_t err, const char *file, int line ) {    
    if (err != hipSuccess) {        
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );        
        exit( EXIT_FAILURE );    
    }
}


template <typename T>
__device__ T warpSum(T val){
    val += __shfl_xor_sync(0xffffffff, val, 16);
    val += __shfl_xor_sync(0xffffffff, val, 8);
    val += __shfl_xor_sync(0xffffffff, val, 4);
    val += __shfl_xor_sync(0xffffffff, val, 2);
    val += __shfl_xor_sync(0xffffffff, val, 1);
    return val;
}

template <typename T>
__device__ T blockSum(T val){
    __shared__ T s_data[SZ / 32];
    int warpIdx = threadIdx.x / 32;
    int laneIdx = threadIdx.x % 32;
    
    val = warpSum(val);
    if(laneIdx == 0) s_data[warpIdx] = val;
    __syncthreads();
    val = (threadIdx.x < 32) ? s_data[laneIdx]:0;
    if(warpIdx == 0) return warpSum(val);
    
}

template <typename T>
__global__ void d_reduceSum(T* d_input,T* d_output, int n){
    T val = 0;
    for(int i = threadIdx.x;i < n;i += blockDim.x){
        val += d_input[i];
    }
    __syncthreads();
    val = blockSum(val);

    if(threadIdx.x == 0){
        
        d_output[0] = val;
    }
    
}

__global__ void printa(float* a, const int num){
    printf("%d: ", threadIdx.x);
    printf("%f\n", a[num -1 - threadIdx.x]);
}

template <typename T>
__global__ void d_reduceSum_v2(T* d_input, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    T val = idx < n ? d_input[idx] : 0;
    // for(int i = threadIdx.x;i < n;i += blockDim.x){
    //     val += d_input[i];
    // }
    // __syncthreads();
    val = blockSum(val);

    if(threadIdx.x == 0){ 
        d_input[blockIdx.x] = val;
    }
    
}

void reduceSum(float* d_input, float* d_output, const int num){
    printa<<<1, 1>>>(d_output, 1);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    d_reduceSum<<<1, SZ>>>(d_input, d_output, num);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());

}

void reduceSum_v2(float* d_input, float* d_output, const int num){
   
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int blockSize = (num + SZ - 1) / SZ;
    d_reduceSum_v2<<<blockSize, SZ>>>(d_input, num);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());
    d_reduceSum<<<1, SZ>>>(d_input, d_output, blockSize);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());

    hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   float elapsedTime=0;
   hipEventElapsedTime(&elapsedTime, start, stop);

   printf("gpu spent time: %f <ms>",  elapsedTime);
   hipEventDestroy(start);
   hipEventDestroy(stop);
    //int blockSize2 = (blockSize + SZ - 1) / SZ;
    //d_reduceSum_v2<<<blockSize2, SZ>>>(d_input, d_input, blockSize);
    //hipDeviceSynchronize();
}

void gpu_data_initial(float* input, float* output, float** d_input, float** d_output, const int num){
    HANDLE_ERROR(hipMalloc((void**)d_input, num * sizeof(float)));
    
    //HANDLE_ERROR(hipMalloc((void**)&d_part, blockSize * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)d_output, 1 * sizeof(float)));
    
    HANDLE_ERROR(hipGetLastError());
}

void cpu_data_to_gpu(float* input, float* d_input, const int num){
    HANDLE_ERROR(hipMemcpy(d_input, input, num * sizeof(float), hipMemcpyHostToDevice));
}

void gpu_data_to_cpu(float* output, float* d_output){
    HANDLE_ERROR(hipMemcpy(output, d_output, 1 * sizeof(float), hipMemcpyDeviceToHost));
}
